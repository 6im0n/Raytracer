#include "hip/hip_runtime.h"
/*
** EPITECH PROJECT, 2024
** kernel.cpp
** File description:
** Create the scene and camera, and render the scene.
*/
#include "../Math/Ray3D.hpp"
#include "IShape.hpp"
#include "IMaterial.hpp"
#include <memory>
#include <hiprand/hiprand_kernel.h>
#include "kernel.cuh"


struct Coord {
    unsigned int x, y;
};

struct RGB {
    unsigned char r, g, b;
};

static Math::Vector3D color(const Math::Ray3D &ray, RayTracer::IShape *scene, int depth)
{
    RayTracer::hits  hit;
    if (scene->hit(ray, 0.001, 10000.0, hit)) {
        Math::Ray3D scattered;
        Math::Vector3D attenuation;
        if (depth < 50 && hit.material->scatter(ray, hit, attenuation, scattered)) {
            return attenuation * color(scattered, scene, depth + 1);
        } else {
            return {0, 0, 0};
        }
    } else {
        Math::Vector3D unitDirection = ray.getDirection().getUnitVector();
        double t = 0.5 * (unitDirection.y + 1.0);
        return Math::Vector3D(1.0, 1.0, 1.0) * (1.0 - t) + Math::Vector3D(0.5, 0.7, 1.0) * t;
    }
}

__global__ void render(unsigned int _axesY_first, unsigned int _axesY_second, int width, int height, int samples, RayTracer::ShapeList* scene, RayTracer::Camera* camera, int x, Coord* config_coord, RGB* config_rgb, int* threadId, hiprandState* state){
    int y = blockIdx.x * blockDim.x + threadIdx.x;
    if (y >= _axesY_first && y < _axesY_second) {
        Math::Vector3D col(0, 0, 0);
        for (int s = 0; s < samples; s++) {
            int idx = threadIdx.x + blockIdx.x * blockDim.x;
            float u = float(x + hiprand_uniform(&state[idx])) / float(width);
            float v = float(y + hiprand_uniform(&state[idx])) / float(height);
            Math::Ray3D ray = camera->getRay(u, v);
            Math::Vector3D p = ray.pointOnRay(2.0);
            col += color(ray, scene, 0);
        }
        col /= float(samples);
        col = Math::Vector3D(sqrt(col.x), sqrt(col.y), sqrt(col.z));
        unsigned char r = (unsigned char) (255.99 * col.x);
        unsigned char g = (unsigned char) (255.99 * col.y);
        unsigned char b = (unsigned char) (255.99 * col.z);
        config_coord[y].x = x;
        config_coord[y].y = y;
        config_rgb[y].r = r;
        config_rgb[y].g = g;
        config_rgb[y].b = b;
    }
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        (*threadId)++;
    }
}